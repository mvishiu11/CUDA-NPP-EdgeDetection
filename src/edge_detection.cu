#include "hip/hip_runtime.h"
#include "edge_detection.hpp"
#include <iostream>
#include <chrono>

/**
 * @brief Kernel to apply Sobel filter to an image.
 * 
 * @param pSrc Input image data.
 * @param pDst Output image data.
 * @param oSizeROI Structure that holds the width and height of the image.
 *
 * @note This kernel is a simple implementation of the Sobel filter. It follows the steps below:
 * 1. Calculate the x and y gradients using the Sobel operator.
 * 2. Compute the gradient magnitude.
 * 3. Normalize the gradient magnitude to the range [0, 255].
**/
__global__ void sobelFilterKernel(const unsigned char* input, unsigned char* output, int width, int height) {
    // Sobel operator kernels
    int Gx[3][3] = {{-1, 0, 1}, {-2, 0, 2}, {-1, 0, 1}};
    int Gy[3][3] = {{-1, -2, -1}, {0, 0, 0}, {1, 2, 1}};
    
    // Get the thread's x and y position in the image
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x > 0 && y > 0 && x < width - 1 && y < height - 1) {
        int sumX = 0;
        int sumY = 0;

        // Apply Sobel filter
        for (int i = -1; i <= 1; ++i) {
            for (int j = -1; j <= 1; ++j) {
                int pixel = input[(y + i) * width + (x + j)];
                sumX += pixel * Gx[i + 1][j + 1];
                sumY += pixel * Gy[i + 1][j + 1];
            }
        }

        // Calculate the gradient magnitude
        int magnitude = min(255, (int)sqrtf(sumX * sumX + sumY * sumY));
        output[y * width + x] = magnitude;
    }
}

void checkNppError(NppStatus status, const char* message) {
    if (status != NPP_SUCCESS) {
        std::cerr << message << ": Error code " << status << std::endl;
        exit(EXIT_FAILURE);
    }
}

void loadImage(const char* filename, Npp8u** pSrc, NppiSize* oSizeROI) {
    cv::Mat h_image = cv::imread(filename, cv::IMREAD_GRAYSCALE);
    
    if (h_image.empty()) {
        std::cerr << "Error: Could not load image " << filename << std::endl;
        exit(EXIT_FAILURE);
    }

    oSizeROI->width = h_image.cols;
    oSizeROI->height = h_image.rows;

    size_t imageSize = oSizeROI->width * oSizeROI->height * sizeof(Npp8u);
    hipMalloc((void**)pSrc, imageSize);
    hipMemcpy(*pSrc, h_image.data, imageSize, hipMemcpyHostToDevice);
}

void saveImage(const char* filename, Npp8u* pDst, NppiSize oSizeROI) {
    cv::Mat h_image(oSizeROI.height, oSizeROI.width, CV_8UC1);
    hipMemcpy(h_image.data, pDst, oSizeROI.width * oSizeROI.height * sizeof(Npp8u), hipMemcpyDeviceToHost);
    cv::imwrite(filename, h_image);
}

void applyCustomSobel(Npp8u* pSrc, Npp8u* pDst, NppiSize oSizeROI) {
    int width = oSizeROI.width;
    int height = oSizeROI.height;

    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, 
                  (height + blockSize.y - 1) / blockSize.y);

    sobelFilterKernel<<<gridSize, blockSize>>>(pSrc, pDst, width, height);

    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void applyNppSobel(Npp8u* pSrc, Npp8u* pDst, NppiSize oSizeROI) {
    NppStatus status = nppiFilterSobelVert_8u_C1R(pSrc, oSizeROI.width, pDst, oSizeROI.width, oSizeROI);
    checkNppError(status, "Failed to apply NPP Sobel filter");
}

void processImage(const std::string& imageFile, const std::string& outputDir, bool useCustom) {
    NppiSize oSizeROI;
    Npp8u* pSrc = nullptr;
    Npp8u* pDst = nullptr;

    loadImage(imageFile.c_str(), &pSrc, &oSizeROI);

    std::cout << "Processing image: " << imageFile << std::endl;
    std::cout << "Image size: " << oSizeROI.width << "x" << oSizeROI.height << std::endl;

    hipMalloc((void**)&pDst, oSizeROI.width * oSizeROI.height * sizeof(Npp8u));

    if (useCustom) {
        applyCustomSobel(pSrc, pDst, oSizeROI);
    } else {
        applyNppSobel(pSrc, pDst, oSizeROI);
    }

    std::string outputFilename = outputDir + "/output_" + (useCustom ? "custom_" : "npp_") + imageFile.substr(imageFile.find_last_of("/") + 1);
    saveImage(outputFilename.c_str(), pDst, oSizeROI);

    hipFree(pSrc);
    hipFree(pDst);
}

void processBatch(const std::string& inputDir, const std::string& outputDir, int batchSize, bool useCustom) {
    std::vector<std::string> imageFiles;
    for (const auto& entry : std::filesystem::directory_iterator(inputDir)) {
        if (entry.is_regular_file()) {
            imageFiles.push_back(entry.path().string());
        }
        if (imageFiles.size() == batchSize) break;
    }

    for (const std::string& imageFile : imageFiles) {
        processImage(imageFile, outputDir, useCustom);
    }
}

int main(int argc, char** argv) {
    if (argc < 3) {
        std::cerr << "Usage: " << argv[0] << " <input_dir> <output_dir>" << std::endl;
        return 1;
    }

    std::string inputDir = argv[1];
    std::string outputDir = argv[2];
    int batchSize = 210;

    // Create separate directories for custom and NPP outputs
    std::string customOutputDir = outputDir + "/output_custom";
    std::string nppOutputDir = outputDir + "/output_npp";
    std::filesystem::create_directories(customOutputDir);
    std::filesystem::create_directories(nppOutputDir);

    std::cout << "|----------------------CUSTOM SOBEL START----------------------|" << std::endl;

    // Benchmark and process custom Sobel
    auto start = std::chrono::high_resolution_clock::now();
    processBatch(inputDir, customOutputDir, batchSize, true);  // true for custom Sobel
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> customDuration = end - start;
    std::cout << "Custom Sobel processing time: " << customDuration.count() << " seconds" << std::endl;

    std::cout << "|----------------------CUSTOM SOBEL END----------------------|" << std::endl;
    std::cout << "|----------------------NPP SOBEL START----------------------|" << std::endl;

    // Benchmark and process NPP Sobel
    start = std::chrono::high_resolution_clock::now();
    processBatch(inputDir, nppOutputDir, batchSize, false);  // false for NPP Sobel
    end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> nppDuration = end - start;
    std::cout << "NPP Sobel processing time: " << nppDuration.count() << " seconds" << std::endl;

    std::cout << "|----------------------NPP SOBEL END----------------------|" << std::endl;

    return 0;
}